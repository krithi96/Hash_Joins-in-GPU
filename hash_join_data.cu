
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<string>
#include<fstream>
#include<iostream>
#include <hiprand/hiprand_kernel.h>
#include <vector>
#include <sstream>



#define p 334214459
#define TABLESIZE 100000
#define maxiterations 10
#define KEYEMPTY -1
#define NOTFOUND -100

__device__
unsigned long long  table[TABLESIZE];



__device__
unsigned long long make_entry(unsigned long key, unsigned long value){
  //printf("key : %d, value : %d",key , value);
  unsigned long long ans = (key<<32)+value;
  //printf ("ans : %d ", (int)ans>>32);
  printf("\n");
  return ans;
}

__device__ unsigned getkey(unsigned long long entry){
return (entry)>>32;
}

__device__ unsigned getvalue(unsigned long long entry){
  return (entry & 0xffffffff) ;
}

__device__
unsigned hash_function_1(unsigned key){
   int a1 = 5;
   int b1 = 2;
   return (((a1*key+b1)%p)%TABLESIZE);
}

__device__
unsigned hash_function_2(unsigned key){
   int a1 = 13;
   int b1 = 7;
   return (((a1*key+b1)%p)%TABLESIZE);
}



__global__
void join(int *Table_B,int *Table_C,int width, int height){
  int temparray[3];
  int index = blockIdx.x * blockDim.x +threadIdx.x;
  unsigned long primkey = Table_B[index*width+0];
  unsigned long value = Table_B[index*width+1];
  unsigned location_1 = hash_function_1(primkey);
  unsigned location_2 = hash_function_2(primkey);
  unsigned long long entry;
  if (getkey(entry = table[location_1])!= primkey)
    if (getkey(entry = table[location_2])!= primkey){
        entry = make_entry(0,NOTFOUND);
    }
  temparray[0]=getkey(entry);
  temparray[1]=getkey(entry);
  temparray[2]=value;
  for(int i =0;i<3;i++){
    printf("%d ",temparray[i]);
  }
  printf("\n");
  __syncthreads();
}

__global__
void hash(int *Table_A, int width, int height){

  int index = blockIdx.x * blockDim.x +threadIdx.x;
  //int stride = blockDim.x * gridDim.x;

   //printf("in the kernel with thread : %d",index);
    unsigned long key = Table_A[index*width+0];
    unsigned long value = Table_A[index*width+1]; //C
    unsigned long long entry = make_entry(key,value);
    //printf("entry: %d",entry);
    unsigned location = hash_function_1(key);
    for (int its = 0; its<maxiterations; its++){
    entry = atomicExch(&table[location], entry);
    key = getkey(entry);
    if (key == 0) {
      printf("threadIdx: %d, table: %llu \n",threadIdx.x,table[location]);
      return;}
    unsigned location1 = hash_function_1(key);
    unsigned location2 = hash_function_2(key);
    if (location == location1)
     location = location2;
    else if (location == location2)
     location = location1;
    };
    printf("chain was too long");
    return ;
}



int main()
{

    int *Table_A;
    int *Table_B;
    int *Table_C;

    int width = 2;
    int height_a = 2500;
    int height_b = 1000000;
    int num1=1;
    int num2 =101;
    int num3 = 201;
    int count =0;

     hipMallocManaged(&Table_A, width * height_a * sizeof(int));
     hipMallocManaged(&Table_B, width * height_b * sizeof(int));
     hipMallocManaged(&Table_C, width * height_b * sizeof(int));
      std::fstream fin;
      fin.open("table_a.csv", std::ios::in);
      std::string line, word;
      int i=0;
      char delimiter;
      int temp[20];
      delimiter = ',';
      //std::string l = "hi how are you";
      while (getline(fin, line,'\n')){
        std::stringstream s(line);
        while (getline(s, word,','))
        {
          Table_A[i]=stoi(word);
          //std::cout<<"table_a: "<<Table_A[i]<<"\n";
          i++;
      }
    }
    fin.close();

    fin.open("table_b.csv", std::ios::in);
    while (getline(fin, line,'\n')){
      std::stringstream s(line);
      while (getline(s, word,','))
      {
        Table_B[i]=stoi(word);
        //std::cout<<"table_b: "<<Table_B[i]<<"\n";
        i++;
      }
  }


    hash<<<3,1024>>>(Table_A, width, height_a);
    join<<<1,1>>>(Table_B,Table_C,width,height_b);
    hipDeviceSynchronize();
    printf("exit ");
}