

#include <hip/hip_runtime.h>
#include<stdio.h>
#include <hiprand/hiprand_kernel.h>


#define p 334214459
#define TABLESIZE 250
#define maxiterations 10
#define KEYEMPTY -1

__device__ 
unsigned long long  table[TABLESIZE];



__device__
unsigned long long make_entry(unsigned long key, unsigned long value){
  //printf("key : %d, value : %d",key , value);
  unsigned long long ans = (key<<32)+value;
  //printf ("ans : %d ", (int)ans>>32);
  printf("\n");
  return ans;
}

__device__ unsigned getkey(unsigned long long entry){
return (entry)>>32;
}

__device__
unsigned hash_function_1(unsigned key){
   int a1 = 5;
   int b1 = 2;
   return (((a1*key+b1)%p)%TABLESIZE);
}

__device__
unsigned hash_function_2(unsigned key){
   int a1 = 13;
   int b1 = 7;
   return (((a1*key+b1)%p)%TABLESIZE);
}


__global__
void hash_join(int *Table_A,int *Table_B, int width, int height){

  int index = blockIdx.x * blockDim.x +threadIdx.x;
  //int stride = blockDim.x * gridDim.x;

  //printf("in the kernel with thread : %d",index);
    unsigned long key = Table_A[index*width+0];
    unsigned long value = Table_A[index*width+1]; //C
    unsigned long long entry = make_entry(key,value);
    //printf("entry: %d",entry);
    unsigned location = hash_function_1(key);
    for (int its = 0; its<maxiterations; its++){
    entry = atomicExch(&table[location], entry);
    key = getkey(entry);
    printf("key, %d \n",key);
    if (key == 0) { 
      printf("return");
      return;}
    unsigned location1 = hash_function_1(key);
    unsigned location2 = hash_function_2(key);
    if (location == location1)
     location = location2;
    else if (location == location2) 
     location = location1;
    };
    printf("chain was too long");
    return ;


   // printf("threadid: %d , table:%lu \n",index,table[location]);
    //__syncthreads();

}


int main()
{
  /* unsigned  key = 1;
   unsigned long long ans =0;
   ans = key<<32;
   printf("ans:  %ld",ans);*/

    
    int *Table_A;
    int *Table_B;
    int *Table_C;

    int width = 2;
    int height = 3;

    hipMallocManaged(&Table_A, width * height * sizeof(int));
    hipMallocManaged(&Table_B, width * height * sizeof(int));
    hipMallocManaged(&Table_C, width * height * sizeof(int));


    Table_A[0]=1;
    Table_A[1]=4;
    Table_A[2]=2;
    Table_A[3]=5;
    Table_A[4]=3; 
    Table_A[5]=6;
    
    Table_B[0]=1;
    Table_B[1]=7;
    Table_B[2]=2;
    Table_B[3]=8;
    Table_B[4]=3; 
    Table_B[5]=9;

    hipMemset(table, KEYEMPTY, TABLESIZE*sizeof(unsigned long long));

    hash_join<<<1,3>>>(Table_A, Table_B, width, height);
    hipDeviceSynchronize();
    printf("exit ");
}